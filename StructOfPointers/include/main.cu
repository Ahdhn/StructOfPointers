#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

#include "helper.h"

struct obj
{
    obj() : ptr0(nullptr), ptr1(nullptr), ptr2(nullptr)
    {
    }    
    int *ptr0, *ptr1, *ptr2;
};

__global__ void kernel(obj my_obj)
{
    printf("\n ptr0= %d, ptr1= %d, ptr2= %d",
           my_obj.ptr0[0],
           my_obj.ptr1[0],
           my_obj.ptr2[0]);
}


int main()
{
    obj my_obj;

    CUDA_ERROR(hipMallocManaged((void**)&my_obj.ptr0, sizeof(int)));
    CUDA_ERROR(hipMallocManaged((void**)&my_obj.ptr1, sizeof(int)));
    CUDA_ERROR(hipMallocManaged((void**)&my_obj.ptr2, sizeof(int)));
    my_obj.ptr0[0] = 42;
    my_obj.ptr1[0] = 77;
    my_obj.ptr2[0] = 99;

    dim3 gridDim(1, 1, 1);
    dim3 blockDim(1, 1, 1);

    void* args[] = {&my_obj};
    CUDA_ERROR(hipLaunchKernel((void*)kernel, gridDim, blockDim, args));

    CUDA_ERROR(hipDeviceSynchronize());
    CUDA_ERROR(hipFree(my_obj.ptr0));
    CUDA_ERROR(hipFree(my_obj.ptr1));
    CUDA_ERROR(hipFree(my_obj.ptr2));
}